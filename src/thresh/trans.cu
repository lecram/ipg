#include "hip/hip_runtime.h"
#include "../ipg.h"
#include "trans.h"

__global__ void
thresh_pixel(unsigned char *img, int width, int height,
             unsigned char threshold,
             unsigned char below, unsigned char above,
             int keep)
{
    int x, y;
    int offset;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    offset = y * width + x;

    if (img[offset] <= threshold) {
        if (!(keep & KEEP_BELOW))
            img[offset] = below;
    }
    else {
        if (!(keep & KEEP_ABOVE))
            img[offset] = above;
    }
}

extern "C"
void
thresh(Matrix *matrix, unsigned char threshold,
       unsigned char below, unsigned char above,
       int keep)
{
    unsigned char *d_img;
    size_t size;
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((matrix->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (matrix->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    size = matrix->width * matrix->height;
    hipMalloc((void **) &d_img, size);
    hipMemcpy(d_img, matrix->data, size, hipMemcpyHostToDevice);
    thresh_pixel<<<numBlocks, threadsPerBlock>>>(
        d_img, matrix->width, matrix->height, threshold, below, above, keep
    );
    hipMemcpy(matrix->data, d_img, size, hipMemcpyDeviceToHost);
    hipFree(d_img);
}
