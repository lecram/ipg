#include "hip/hip_runtime.h"
#include "../ipg.h"
#include "trans.h"

__global__ void
thresh_pixel(unsigned char *img, int width, int height,
             unsigned char threshold,
             unsigned char below, unsigned char above,
             int keep)
{
    int x, y;
    int offset;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    offset = y * width + x;

    if (img[offset] <= threshold) {
        if (!(keep & KEEP_BELOW))
            img[offset] = below;
    }
    else {
        if (!(keep & KEEP_ABOVE))
            img[offset] = above;
    }
}

extern "C"
void
thresh(Matrix *matrix, unsigned char threshold,
       unsigned char below, unsigned char above,
       int keep)
{
    unsigned char *d_img;
    size_t size;
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((matrix->width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (matrix->height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    size = matrix->width * matrix->height;
    hipMalloc((void **) &d_img, size);
    hipMemcpy(d_img, matrix->data, size, hipMemcpyHostToDevice);
    thresh_pixel<<<numBlocks, threadsPerBlock>>>(
        d_img, matrix->width, matrix->height, threshold, below, above, keep
    );
    hipMemcpy(matrix->data, d_img, size, hipMemcpyDeviceToHost);
    hipFree(d_img);
}

__global__ void
thresh_pixelN(unsigned char *img, int width, int height,
              unsigned char threshold,
              unsigned char below, unsigned char above,
              int keep)
{
    int x, y, z;
    int offset;
    int size;

    size = width * height;
    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    offset = y * width + x;
    z = offset / size;
    x %= width;
    y %= height;
    if (z >= N)
        return;


    if (img[offset] <= threshold) {
        if (!(keep & KEEP_BELOW))
            img[offset] = below;
    }
    else {
        if (!(keep & KEEP_ABOVE))
            img[offset] = above;
    }
}

extern "C"
void
threshN(Matrix *matrix, unsigned char threshold,
        unsigned char below, unsigned char above,
        int keep)
{
    unsigned char *d_img;
    size_t size;
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((matrix->width * N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (matrix->height * N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    size = matrix->width * matrix->height * N;
    hipMalloc((void **) &d_img, size);
    hipMemcpy(d_img, matrix->data, size, hipMemcpyHostToDevice);
    thresh_pixelN<<<numBlocks, threadsPerBlock>>>(
        d_img, matrix->width, matrix->height, threshold, below, above, keep
    );
    hipMemcpy(matrix->data, d_img, size, hipMemcpyDeviceToHost);
    hipFree(d_img);
}
